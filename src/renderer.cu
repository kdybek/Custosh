#include "hip/hip_runtime.h"
#include "renderer.h"

#include "internal/debug_macros.h"
#include "internal/gpu_memory.h"
#include "internal/windows_console_screen_buffer.h"

namespace Custosh::Renderer
{
    namespace
    {
        /* Auxiliary structs */
        struct triangle3D_t
        {
            Vertex3D p0;
            Vertex3D p1;
            Vertex3D p2;

            __host__ __device__ explicit triangle3D_t(
                    const Vertex3D& p0 = Vertex3D(),
                    const Vertex3D& p1 = Vertex3D(),
                    const Vertex3D& p2 = Vertex3D()
            ) : p0(p0), p1(p1), p2(p2)
            {
            }
        };

        struct triangle2D_t
        {
            Vertex2D p0;
            Vertex2D p1;
            Vertex2D p2;

            __host__ __device__ explicit triangle2D_t(
                    const Vertex2D& p0 = Vertex2D(),
                    const Vertex2D& p1 = Vertex2D(),
                    const Vertex2D& p2 = Vertex2D()
            ) : p0(p0), p1(p1), p2(p2)
            {
            }
        };

        struct boundingBox_t
        {
            float xMax;
            float xMin;
            float yMax;
            float yMin;

            __host__ __device__ explicit boundingBox_t(
                    float xMax = 0.f,
                    float xMin = 0.f,
                    float yMax = 0.f,
                    float yMin = 0.f
            ) : xMax(xMax), xMin(xMin), yMax(yMax), yMin(yMin)
            {
            }
        };

        struct barycentricCoords_t
        {
            float alpha;
            float beta;
            float gamma;

            __host__ __device__ explicit barycentricCoords_t(
                    float alpha = 0.f,
                    float beta = 0.f,
                    float gamma = 0.f
            ) : alpha(alpha), beta(beta), gamma(gamma)
            {
            }
        };

        struct fragment_t
        {
            bool occupied;
            Vertex3D coords;
            Vector3<float> normal;

            __host__ __device__ explicit fragment_t(
                    bool occupied = false,
                    const Vertex3D& coords = Vertex3D(),
                    const Vector3<float>& normal = Vector3<float>()
            ) : occupied(occupied), coords(coords), normal(normal)
            {
            }
        };

        struct triangleVariables_t
        {
            float area2x;
            boundingBox_t boundingBox;
            Vector3<float> normal;

            __host__ __device__ explicit triangleVariables_t(
                    float area2x = 0.f,
                    boundingBox_t boundingBox = boundingBox_t(),
                    Vector3<float> normal = Vector3<float>()
            ) : area2x(area2x), boundingBox(boundingBox), normal(normal)
            {
            }
        };

        struct fullTriangleInfo_t
        {
            triangle3D_t coords3D;
            triangle2D_t coords2D;
            triangleVariables_t triangleVars;

            __host__ __device__ explicit fullTriangleInfo_t(
                    triangle3D_t coords3D = triangle3D_t(),
                    triangle2D_t coords2D = triangle2D_t(),
                    triangleVariables_t triangleVars = triangleVariables_t()
            ) : coords3D(coords3D), coords2D(coords2D), triangleVars(triangleVars)
            {
            }
        };

        /* Host constants */
        constexpr unsigned int H_BASE_DEV_WSPACE_SIZE = 8;
        constexpr Vertex3D H_CCV_MIN_CORNER = {-1.f, -1.f, -1.f};
        constexpr Vertex3D H_CCV_MAX_CORNER = {1.f, 1.f, 1.f};
        constexpr float H_PM_NEAR_PLANE = 1.f;
        constexpr float H_PM_FAR_PLANE = 1000.f;
        constexpr unsigned int H_THREADS_PER_BLOCK_X = 8;
        constexpr unsigned int H_THREADS_PER_BLOCK_Y = 8;
        constexpr unsigned int H_THREADS_PER_BLOCK = H_THREADS_PER_BLOCK_X * H_THREADS_PER_BLOCK_Y;
        constexpr TransformMatrix H_IDENTITY_MATRIX = {{1.f, 0.f, 0.f, 0.f},
                                                       {0.f, 1.f, 0.f, 0.f},
                                                       {0.f, 0.f, 1.f, 0.f},
                                                       {0.f, 0.f, 0.f, 1.f}};

        /* Device constants */
        __constant__ constexpr char D_ASCII_BY_BRIGHTNESS[93] =
                R"( `.-':_,^=;><+!rc*/z?sLTv)J7(|Fi{C}fI31tlu[neoZ5Yxjya]2ESwqkP6h9d4VpOGbUAKXHm8RD#$Bg0MNWQ%&@)";
        __constant__ constexpr unsigned int D_NUM_ASCII = 92;
        __constant__ constexpr unsigned int D_THREADS_PER_BLOCK = H_THREADS_PER_BLOCK;

        /* Device global variables */
        __constant__ constinit lightSource_t g_devLightSource;

        /* @formatter:off */

        /* Host global variables */
        WindowsConsoleScreenBuffer& getFrontBuffer() { static WindowsConsoleScreenBuffer s_frontBuffer; return s_frontBuffer; }
        WindowsConsoleScreenBuffer& getBackBuffer() { static WindowsConsoleScreenBuffer s_backBuffer; return s_backBuffer; }
        HostPtr<TransformMatrix>& getTransformHostPtr() { static HostPtr<TransformMatrix> s_transformHostPtr(H_BASE_DEV_WSPACE_SIZE); return s_transformHostPtr; }
        HostPtr<char>& getFrameBufferHostPtr() { static HostPtr<char> s_frameBufferHostPtr(H_BASE_DEV_WSPACE_SIZE); return s_frameBufferHostPtr; }

        /* Device working space pointers */
        DevPtr<meshVertex_t>& getMeshVertexDevPtr() { static DevPtr<meshVertex_t> s_meshVectorDevPtr(H_BASE_DEV_WSPACE_SIZE); return s_meshVectorDevPtr; }
        DevPtr<triangleIndices_t>& getTriangleIndDevPtr() { static DevPtr<triangleIndices_t> s_triangleIndDevPtr(H_BASE_DEV_WSPACE_SIZE); return s_triangleIndDevPtr; }
        DevPtr<TransformMatrix>& getTransformDevPtr() { static DevPtr<TransformMatrix> s_transformDevPtr(H_BASE_DEV_WSPACE_SIZE); return s_transformDevPtr; }
        DevPtr<Vertex2D>& getVertex2DDevPtr() { static DevPtr<Vertex2D> s_vertex2DDevPtr(H_BASE_DEV_WSPACE_SIZE); return s_vertex2DDevPtr; }
        DevPtr<triangleVariables_t>& getTriangleVarsPtr() { static DevPtr<triangleVariables_t> s_triangleVarsPtr(H_BASE_DEV_WSPACE_SIZE); return s_triangleVarsPtr; }
        DevPtr<char>& getFrameBufferDevPtr() { static DevPtr<char> s_frameBufferDevPtr(H_BASE_DEV_WSPACE_SIZE); return s_frameBufferDevPtr; }

        /* @formatter:on */

        /* Device auxiliary functions */
        template<typename T>
        [[nodiscard]] __device__ T clamp(T a, T min, T max)
        {
            if (a < min) { return min; }
            else if (a > max) { return max; }
            else { return a; }
        }

        template<typename T>
        [[nodiscard]] __device__ T max3(T a, T b, T c)
        {
            return max(max(a, b), c);
        }

        template<typename T>
        [[nodiscard]] __device__ T min3(T a, T b, T c)
        {
            return min(min(a, b), c);
        }

        template<typename T>
        __device__ void swap(T& a, T& b)
        {
            T temp = a;
            a = b;
            b = temp;
        }

        [[nodiscard]] __device__ char brightnessToASCII(float brightness)
        {
            unsigned int idx = ceil(brightness * static_cast<float>(D_NUM_ASCII - 1));
            return D_ASCII_BY_BRIGHTNESS[idx];
        }

        [[nodiscard]] __device__ boundingBox_t findBounds(const triangle2D_t& triangle2D)
        {
            boundingBox_t boundingBox;

            boundingBox.xMax = max3(ceil(triangle2D.p0.x()),
                                    ceil(triangle2D.p1.x()),
                                    ceil(triangle2D.p2.x()));
            boundingBox.xMin = min3(floor(triangle2D.p0.x()),
                                    floor(triangle2D.p1.x()),
                                    floor(triangle2D.p2.x()));
            boundingBox.yMax = max3(ceil(triangle2D.p0.y()),
                                    ceil(triangle2D.p1.y()),
                                    ceil(triangle2D.p2.y()));
            boundingBox.yMin = min3(floor(triangle2D.p0.y()),
                                    floor(triangle2D.p1.y()),
                                    floor(triangle2D.p2.y()));

            return boundingBox;
        }

        [[nodiscard]] __device__ bool isBottomOrRight(const Vertex2D& a,
                                                      const Vertex2D& b)
        {
            auto edge = Vector2<float>(b - a);
            bool bottomEdge = edge.y() == 0 && edge.x() < 0;
            bool rightEdge = edge.y() < 0;

            return bottomEdge || rightEdge;
        }

        [[nodiscard]] __device__ float cross2D(const Vertex2D& a,
                                               const Vertex2D& b,
                                               const Vertex2D& c)
        {
            auto ab = Vector2<float>(b - a);
            auto ac = Vector2<float>(c - a);

            return ab.x() * ac.y() - ab.y() * ac.x();
        }

        [[nodiscard]] __device__ bool inBoundingBox(const boundingBox_t& boundingBox,
                                                    const Vertex2D& p)
        {
            if (p.x() >= boundingBox.xMin && p.x() <= boundingBox.xMax &&
                p.y() >= boundingBox.yMin && p.y() <= boundingBox.yMax) {
                return true;
            }
            else { return false; }
        }

        [[nodiscard]] __device__ bool inTriangle(const triangle2D_t& triangle2D,
                                                 const boundingBox_t& boundingBox,
                                                 const Vertex2D& p,
                                                 float triangleArea2x,
                                                 barycentricCoords_t& barycentricCoords)
        {
            if (!inBoundingBox(boundingBox, p)) { return false; }

            float w0 = cross2D(triangle2D.p1, triangle2D.p2, p);
            float w1 = cross2D(triangle2D.p2, triangle2D.p0, p);
            float w2 = cross2D(triangle2D.p0, triangle2D.p1, p);

            if (w0 == 0.f && isBottomOrRight(triangle2D.p1, triangle2D.p2)) { return false; }
            if (w1 == 0.f && isBottomOrRight(triangle2D.p2, triangle2D.p0)) { return false; }
            if (w2 == 0.f && isBottomOrRight(triangle2D.p0, triangle2D.p1)) { return false; }

            barycentricCoords.alpha = w0 / triangleArea2x;
            barycentricCoords.beta = w1 / triangleArea2x;
            barycentricCoords.gamma = w2 / triangleArea2x;

            return (w0 >= 0.f && w1 >= 0.f && w2 >= 0.f);
        }

        [[nodiscard]] __device__ Vertex3D getCartesianCoords(const triangle3D_t& triangle3D,
                                                             const barycentricCoords_t& bc)
        {
            return {triangle3D.p0.x() * bc.alpha + triangle3D.p1.x() * bc.beta + triangle3D.p2.x() * bc.gamma,
                    triangle3D.p0.y() * bc.alpha + triangle3D.p1.y() * bc.beta + triangle3D.p2.y() * bc.gamma,
                    triangle3D.p0.z() * bc.alpha + triangle3D.p1.z() * bc.beta + triangle3D.p2.z() * bc.gamma};
        }

        [[nodiscard]] __device__ float distanceSq(const Vertex3D& a, const Vertex3D& b)
        {
            return static_cast<float>(pow((a.x() - b.x()), 2) + pow((a.y() - b.y()), 2) + pow((a.z() - b.z()), 2));
        }

        [[nodiscard]] __device__ float fragmentBrightness(const fragment_t& p, const lightSource_t& ls)
        {
            if (!p.occupied) { return 0.f; }

            float distSq = distanceSq(p.coords, ls.coords);
            auto pointToLightSourceVec = Vector3<float>(ls.coords - p.coords);
            auto pointToLightSourceVecNorm = Vector3<float>(pointToLightSourceVec.normalized());
            float cos = pointToLightSourceVecNorm.dot(p.normal);

            return clamp(max(cos, 0.f) * ls.intensity / distSq, 0.f, 1.f);
        }

        // The vertices are in clockwise order, but we're looking from 0 towards positive z values.
        [[nodiscard]] __device__ Vector3<float> triangleNormal(const triangle3D_t& triangle3D)
        {
            Vector3<float> normal = Vector3<float>(triangle3D.p2 - triangle3D.p0).cross(
                    Vector3<float>(triangle3D.p1 - triangle3D.p0));

            return Vector3<float>(normal.normalized());
        }

        [[nodiscard]] __device__ triangle2D_t getTriangle2D(const triangleIndices_t& triangleIndices,
                                                            const Vertex2D* vertex2DPtr)
        {
            return triangle2D_t(vertex2DPtr[triangleIndices.p0],
                                vertex2DPtr[triangleIndices.p1],
                                vertex2DPtr[triangleIndices.p2]);
        }

        [[nodiscard]] __device__ triangle3D_t getTriangle3D(const triangleIndices_t& triangleIndices,
                                                            const meshVertex_t* meshVertexPtr)
        {
            return triangle3D_t(meshVertexPtr[triangleIndices.p0].coords,
                                meshVertexPtr[triangleIndices.p1].coords,
                                meshVertexPtr[triangleIndices.p2].coords);
        }

        /* Kernels */
        __global__ void vertexShader(meshVertex_t* meshVertexPtr,
                                     unsigned int numVertices,
                                     const TransformMatrix* transformMatPtr,
                                     PerspectiveProjectionMatrix ppm,
                                     Vertex2D* vertex2DPtr)
        {
            const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

            if (i >= numVertices) { return; }

            meshVertex_t meshVertex = meshVertexPtr[i];

            Vector4<float> updatedVertex4D = Vector4<float>(transformMatPtr[meshVertex.meshIdx] *
                                                            meshVertex.coords.toHomogeneous()).normalizeW();

            Vector4<float> vertex4DPerspective = Vector4<float>(ppm * updatedVertex4D).normalizeW();

            meshVertexPtr[i] = meshVertex_t({updatedVertex4D.x(), updatedVertex4D.y(), updatedVertex4D.z()},
                                            meshVertex.meshIdx);

            vertex2DPtr[i] = {vertex4DPerspective.x(), vertex4DPerspective.y()};
        }

        __global__ void geometryShader(const triangleIndices_t* triangleIndPtr,
                                       unsigned int numTriangles,
                                       const Vertex2D* vertex2DPtr,
                                       const meshVertex_t* meshVertexPtr,
                                       triangleVariables_t* triangleVarsPtr)
        {
            const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

            if (i >= numTriangles) { return; }

            triangleIndices_t tIndRef = triangleIndPtr[i];
            auto& tVarsRef = triangleVarsPtr[i];

            triangle2D_t triangle2D = getTriangle2D(tIndRef, vertex2DPtr);
            triangle3D_t triangle3D = getTriangle3D(tIndRef, meshVertexPtr);

            tVarsRef.area2x = cross2D(triangle2D.p0, triangle2D.p1, triangle2D.p2);
            tVarsRef.normal = triangleNormal(triangle3D);
            tVarsRef.boundingBox = findBounds(triangle2D);
        }

        __global__ void fragmentShader(unsigned int rows,
                                       unsigned int cols,
                                       const triangleIndices_t* indexPtr,
                                       unsigned int numTriangles,
                                       const Vertex2D* vertex2DPtr,
                                       const meshVertex_t* meshVertexPtr,
                                       const triangleVariables_t* tVarsPtr,
                                       char* frameBuffer)
        {
            __shared__ char sh_memory[D_THREADS_PER_BLOCK * sizeof(fullTriangleInfo_t)];

            auto* sh_triangleInfoPtr = reinterpret_cast<fullTriangleInfo_t*>(sh_memory);

            const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
            const unsigned int sharedMemIdx = threadIdx.x * blockDim.y + threadIdx.y;
            const unsigned int blocksToLoad = (numTriangles + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;

            fragment_t fragment;

            for (unsigned int i = 0; i < blocksToLoad; ++i) {
                const unsigned int globalMemIdx = i * D_THREADS_PER_BLOCK + sharedMemIdx;

                if (globalMemIdx < numTriangles) {
                    triangleIndices_t triangleIndices = indexPtr[globalMemIdx];
                    auto& sh_tInfoRef = sh_triangleInfoPtr[sharedMemIdx];

                    sh_tInfoRef.coords3D = getTriangle3D(triangleIndices, meshVertexPtr);
                    sh_tInfoRef.coords2D = getTriangle2D(triangleIndices, vertex2DPtr);
                    sh_tInfoRef.triangleVars = tVarsPtr[globalMemIdx];
                }

                __syncthreads();

                if (y < rows && x < cols) {
                    const unsigned int trianglesInBlock = min(numTriangles - i * D_THREADS_PER_BLOCK,
                                                              D_THREADS_PER_BLOCK);

                    for (unsigned int j = 0; j < trianglesInBlock; ++j) {
                        fullTriangleInfo_t tInfo = sh_triangleInfoPtr[j];
                        barycentricCoords_t bc;

                        if (tInfo.triangleVars.area2x == 0.f) { continue; }

                        if (inTriangle(tInfo.coords2D,
                                       tInfo.triangleVars.boundingBox,
                                       Vertex2D({static_cast<float>(x), static_cast<float>(y)}),
                                       tInfo.triangleVars.area2x,
                                       bc)) {
                            Vertex3D projectedPoint = getCartesianCoords(tInfo.coords3D, bc);

                            if (!fragment.occupied || fragment.coords.z() > projectedPoint.z()) {
                                fragment.occupied = true;
                                fragment.coords = projectedPoint;
                                fragment.normal = tInfo.triangleVars.normal;
                            }
                        }
                    }
                }

                __syncthreads();
            }

            if (y < rows && x < cols) {
                frameBuffer[y * cols + x] = brightnessToASCII(fragmentBrightness(fragment, g_devLightSource));
            }
        }

        /* Host auxiliary functions */
        [[nodiscard]] __host__ PerspectiveProjectionMatrix CCV2ScreenPPM(unsigned int screenRows,
                                                                         unsigned int screenCols)
        {
            return {PerspectiveMatrix(H_PM_NEAR_PLANE, H_PM_FAR_PLANE),
                    OrthographicProjectionMatrix(H_CCV_MIN_CORNER,
                                                 H_CCV_MAX_CORNER,
                                                 {0.f, 0.f, 0.f},
                                                 {static_cast<float>(screenCols), static_cast<float>(screenRows),
                                                  0.f})};
        }

        __host__ void resizeSceneDependentPtrs(unsigned int numVertices,
                                               unsigned int numTriangles,
                                               unsigned int numMeshes)
        {
            getMeshVertexDevPtr().resizeAndDiscardData(numVertices);
            getVertex2DDevPtr().resizeAndDiscardData(numVertices);

            getTriangleIndDevPtr().resizeAndDiscardData(numTriangles);
            getTriangleVarsPtr().resizeAndDiscardData(numTriangles);

            getTransformDevPtr().resizeAndDiscardData(numMeshes);
            getTransformHostPtr().resizeAndDiscardData(numMeshes);
        }

        __host__ void resizeScreenDependentPtrs(unsigned int screenRows,
                                                unsigned int screenCols)
        {
            getFrameBufferHostPtr().resizeAndDiscardData(screenRows * screenCols);
            getFrameBufferDevPtr().resizeAndDiscardData(screenRows * screenCols);
        }

        __host__ void callVertexShader(const PerspectiveProjectionMatrix& ppm)
        {
            unsigned int numVertices = getMeshVertexDevPtr().size();

            unsigned int threadsPerBlock = std::min(numVertices, static_cast<unsigned int>(H_THREADS_PER_BLOCK));
            unsigned int numBlocks = (numVertices + threadsPerBlock - 1) / threadsPerBlock;

            vertexShader<<<numBlocks, threadsPerBlock>>>(getMeshVertexDevPtr().get(),
                                                         numVertices,
                                                         getTransformDevPtr().get(),
                                                         ppm,
                                                         getVertex2DDevPtr().get());
            CUSTOSH_CUDA_CHECK(hipGetLastError());
        }

        __host__ void callGeometryShader()
        {
            unsigned int numTriangles = getTriangleIndDevPtr().size();

            unsigned int threadsPerBlock = std::min(numTriangles, H_THREADS_PER_BLOCK);
            unsigned int numBlocks = (numTriangles + threadsPerBlock - 1) / threadsPerBlock;

            geometryShader<<<numBlocks, threadsPerBlock>>>(getTriangleIndDevPtr().get(),
                                                           numTriangles,
                                                           getVertex2DDevPtr().get(),
                                                           getMeshVertexDevPtr().get(),
                                                           getTriangleVarsPtr().get());
            CUSTOSH_CUDA_CHECK(hipGetLastError());
        }

        __host__ void callFragmentShader(unsigned int screenRows,
                                         unsigned int screenCols)
        {
            unsigned int numTriangles = getTriangleIndDevPtr().size();

            dim3 threadsPerBlock(H_THREADS_PER_BLOCK_X, H_THREADS_PER_BLOCK_Y);
            dim3 numBlocks((screenCols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (screenRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

            fragmentShader<<<numBlocks, threadsPerBlock>>>(screenRows,
                                                           screenCols,
                                                           getTriangleIndDevPtr().get(),
                                                           numTriangles,
                                                           getVertex2DDevPtr().get(),
                                                           getMeshVertexDevPtr().get(),
                                                           getTriangleVarsPtr().get(),
                                                           getFrameBufferDevPtr().get());
            CUSTOSH_CUDA_CHECK(hipGetLastError());
        }

        __host__ void resetTransformMatrices()
        {
            for (unsigned int i = 0; i < getTransformHostPtr().size(); ++i) {
                getTransformHostPtr().get()[i] = H_IDENTITY_MATRIX;
            }

            getTransformHostPtr().loadToDev(getTransformDevPtr().get(), getTransformHostPtr().size());
        }

        __host__ void setLightSource(const lightSource_t& ls)
        {
            CUSTOSH_CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_devLightSource), &ls, sizeof(lightSource_t)));
        }

        __host__ void renderingPipeline(unsigned int screenRows,
                                        unsigned int screenCols,
                                        const PerspectiveProjectionMatrix& ppm)
        {
            callVertexShader(ppm);

            CUSTOSH_CUDA_CHECK(hipDeviceSynchronize());

            callGeometryShader();

            CUSTOSH_CUDA_CHECK(hipDeviceSynchronize());

            callFragmentShader(screenRows, screenCols);

            // Host and device will synchronize when copying the frame buffer to host,
            // but I synchronize them here to accurately measure time.
            CUSTOSH_DEBUG_CALL(CUSTOSH_CUDA_CHECK(hipDeviceSynchronize()));
        }

        __host__ void drawFrameBuffer(unsigned int screenRows,
                                      unsigned int screenCols)
        {
            getBackBuffer().draw(getFrameBufferHostPtr().get(), screenRows, screenCols);
            getBackBuffer().activate();
            std::swap(getFrontBuffer(), getBackBuffer());
        }

    } // anonymous

    __host__ void loadScene(const Scene& scene)
    {
        resizeSceneDependentPtrs(scene.numVertices(), scene.numTriangles(), scene.numMeshes());

        scene.loadVerticesToDev(getMeshVertexDevPtr().get());
        scene.loadTrianglesToDev(getTriangleIndDevPtr().get());

        resetTransformMatrices();

        setLightSource(scene.lightSource());
    }

    __host__ void loadTransformMatrix(const TransformMatrix& tm, unsigned int meshIdx)
    {
        if (meshIdx >= getTransformHostPtr().size()) { throw CustoshException("invalid mesh index"); }

        getTransformHostPtr().get()[meshIdx] = tm;

        getTransformHostPtr().loadToDev(getTransformDevPtr().get(), getTransformHostPtr().size());
    }

    __host__ void transformVerticesAndDraw()
    {
        Vector2<unsigned int> screenDim = getBackBuffer().getWindowDimensions();

        screenDim.x() = std::min(screenDim.x(), screenDim.y());
        screenDim.y() = std::min(screenDim.x(), screenDim.y());

        if (screenDim.x() == 0 || screenDim.y() == 0) { return; }

        PerspectiveProjectionMatrix ppm = CCV2ScreenPPM(screenDim.y(), screenDim.x());

        resizeScreenDependentPtrs(screenDim.y(), screenDim.x());

        CUSTOSH_DEBUG_LOG_TIME(renderingPipeline(screenDim.y(), screenDim.x(), ppm), "rendering pipeline");

        // This is slow, but if I want to draw in the terminal there is no way to bypass the CPU as far as I know.
        CUSTOSH_DEBUG_LOG_TIME(
                getFrameBufferDevPtr().loadToHost(getFrameBufferHostPtr().get(), screenDim.y() * screenDim.x()),
                "fetch frame buffer from GPU");

        // This is currently the biggest bottleneck, but works fine most of the time.
        CUSTOSH_DEBUG_LOG_TIME(drawFrameBuffer(screenDim.y(), screenDim.x()), "write to terminal");
    }

} // Custosh::Renderer