#include "hip/hip_runtime.h"
#include "Renderer.cuh"

#include "WindowsConsoleScreenBuffer.h"

#define BASE_DEV_WSPACE_SIZE 8
#define CCV_MIN_CORNER {-1.f, -1.f, -1.f}
#define CCV_MAX_CORNER {1.f, 1.f, 1.f}
#define PM_NEAR_PLANE 1.f
#define PM_FAR_PLANE 10.f
#define MAX_THREADS_PER_BLOCK 256
#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

namespace Custosh::Renderer
{
    namespace
    {
        /* Device global variables */
        __constant__ const char* g_devASCIIByBrightness =
                R"( `.-':_,^=;><+!rc*/z?sLTv)J7(|Fi{C}fI31tlu[neoZ5Yxjya]2ESwqkP6h9d4VpOGbUAKXHm8RD#$Bg0MNWQ%&@)";
        __constant__ const unsigned int g_devNumASCII = 94; // TODO: make sure it's right

        /* Host global variables */
        WindowsConsoleScreenBuffer g_hostActiveBuf;
        WindowsConsoleScreenBuffer g_hostInactiveBuf;
        HostPtr<char> g_hostCharHostPtr(BASE_DEV_WSPACE_SIZE);

        /* Device working space pointers */
        DevPtr<Vertex3D> g_hostVertex3DDevPtr(BASE_DEV_WSPACE_SIZE);
        DevPtr<triangleIndices_t> g_hostIndexDevPtr(BASE_DEV_WSPACE_SIZE);
        DevPtr<Vertex2D> g_hostVertex2DDevPtr(BASE_DEV_WSPACE_SIZE);
        DevPtr<boundingBox_t> g_hostBoundingBoxDevPtr(BASE_DEV_WSPACE_SIZE);
        DevPtr<float> g_hostTriangleCross2DDevPtr(BASE_DEV_WSPACE_SIZE);
        DevPtr<Vector3<float>> g_hostTriangleNormalDevPtr(BASE_DEV_WSPACE_SIZE);
        DevPtr<char> g_hostCharDevPtr(BASE_DEV_WSPACE_SIZE);

        /* Device auxiliary functions */
        [[nodiscard]] __device__ char brightnessToASCII(float brightness)
        {
            unsigned int idx = ceil(brightness * static_cast<float>(g_devNumASCII - 1));
            return g_devASCIIByBrightness[idx];
        }

        [[nodiscard]] __device__ boundingBox_t findBounds(const triangle2D_t& triangle2D)
        {
            boundingBox_t boundingBox;

            boundingBox.xMax = max3(ceil(triangle2D.p0.x()),
                                    ceil(triangle2D.p1.x()),
                                    ceil(triangle2D.p2.x()));
            boundingBox.xMin = min3(floor(triangle2D.p0.x()),
                                    floor(triangle2D.p1.x()),
                                    floor(triangle2D.p2.x()));
            boundingBox.yMax = max3(ceil(triangle2D.p0.y()),
                                    ceil(triangle2D.p1.y()),
                                    ceil(triangle2D.p2.y()));
            boundingBox.yMin = min3(floor(triangle2D.p0.y()),
                                    floor(triangle2D.p1.y()),
                                    floor(triangle2D.p2.y()));

            return boundingBox;
        }

        [[nodiscard]] __device__ bool isBottomOrRight(const Vertex2D& a,
                                                      const Vertex2D& b)
        {
            auto edge = Vector2<float>(b - a);
            bool bottomEdge = edge.y() == 0 && edge.x() < 0;
            bool rightEdge = edge.y() < 0;

            return bottomEdge || rightEdge;
        }

        [[nodiscard]] __device__ float cross2D(const Vertex2D& a,
                                               const Vertex2D& b,
                                               const Vertex2D& c)
        {
            auto ab = Vector2<float>(b - a);
            auto ac = Vector2<float>(c - a);

            return ab.x() * ac.y() - ab.y() * ac.x();
        }

        [[nodiscard]] __device__ bool inBoundingBox(const boundingBox_t& boundingBox,
                                                    const Vertex2D& p)
        {
            if (p.x() >= boundingBox.xMin && p.x() <= boundingBox.xMax &&
                p.y() >= boundingBox.yMin && p.y() <= boundingBox.yMax) {
                return true;
            }
            else { return false; }
        }

        [[nodiscard]] __device__ bool inTriangle(const triangle2D_t& triangle2D,
                                                 const boundingBox_t& boundingBox,
                                                 const Vertex2D& p,
                                                 float triangleArea2x,
                                                 barycentricCoords_t& barycentricCoords)
        {
            if (!inBoundingBox(boundingBox, p)) { return false; }

            float w0 = cross2D(triangle2D.p1, p, triangle2D.p2);
            float w1 = cross2D(triangle2D.p2, p, triangle2D.p0);
            float w2 = cross2D(triangle2D.p0, p, triangle2D.p1);

            if (w0 == 0 && isBottomOrRight(triangle2D.p1, triangle2D.p2)) { return false; }
            if (w1 == 0 && isBottomOrRight(triangle2D.p2, triangle2D.p0)) { return false; }
            if (w2 == 0 && isBottomOrRight(triangle2D.p0, triangle2D.p1)) { return false; }

            barycentricCoords.alpha = w0 / triangleArea2x;
            barycentricCoords.beta = w1 / triangleArea2x;
            barycentricCoords.gamma = w2 / triangleArea2x;

            return (w0 >= 0.f && w1 >= 0.f && w2 >= 0.f);
        }

        [[nodiscard]] __device__ Vertex2D applyPerspective(const Vertex3D& p,
                                                           const PerspectiveProjMatrix& ppm)
        {
            Vector4<float> pPerspective = Vector4<float>(ppm * p.toHomogeneous()).normalizeW();
            return {pPerspective.x(), pPerspective.y()};
        }

        [[nodiscard]] __device__ Vertex3D getCartesianCoords(const triangle3D_t& triangle3D,
                                                             const barycentricCoords_t& bc)
        {
            return {triangle3D.p0.x() * bc.alpha + triangle3D.p1.x() * bc.beta + triangle3D.p2.x() * bc.gamma,
                    triangle3D.p0.y() * bc.alpha + triangle3D.p1.y() * bc.beta + triangle3D.p2.y() * bc.gamma,
                    triangle3D.p0.z() * bc.alpha + triangle3D.p1.z() * bc.beta + triangle3D.p2.z() * bc.gamma};
        }

        [[nodiscard]] __device__ float distanceSq(const Vertex3D& a, const Vertex3D& b)
        {
            return static_cast<float>(pow((a.x() - b.x()), 2) + pow((a.y() - b.y()), 2) + pow((a.z() - b.z()), 2));
        }

        [[nodiscard]] __device__ float pointBrightness(const fragment_t& p, const lightSource_t& ls)
        {
            float distSq = distanceSq(p.coords, ls.coords);
            auto pointToLightSourceVec = Vector3<float>(ls.coords - p.coords);
            auto pointToLightSourceVecNorm = Vector3<float>(pointToLightSourceVec.normalized());
            float cos = pointToLightSourceVecNorm.dot(p.normal);

            return clamp(max(cos, 0.f) * ls.intensity / distSq, 0.f, 1.f);
        }

        // The vertices are clockwise oriented, but we're looking from 0 towards positive z values.
        [[nodiscard]] __device__ Vector3<float> triangleNormal(const triangle3D_t& triangle3D)
        {
            Vector3<float> normal = Vector3<float>(triangle3D.p1 - triangle3D.p0).cross(
                    Vector3<float>(triangle3D.p2 - triangle3D.p0));

            return Vector3<float>(normal.normalized());
        }

        [[nodiscard]] __device__ triangle2D_t getTriangle2D(const triangleIndices_t& triangleIndices,
                                                            const Vertex2D* vertex2DPtr)
        {
            return triangle2D_t(vertex2DPtr[triangleIndices.p0],
                                vertex2DPtr[triangleIndices.p1],
                                vertex2DPtr[triangleIndices.p2]);
        }

        [[nodiscard]] __device__ triangle3D_t getTriangle3D(const triangleIndices_t& triangleIndices,
                                                            const Vertex3D* vertex3DPtr)
        {
            return triangle3D_t(vertex3DPtr[triangleIndices.p0],
                                vertex3DPtr[triangleIndices.p1],
                                vertex3DPtr[triangleIndices.p2]);
        }

        /* Kernels */
        // TODO: vertex manipulation (translation, rotation, etc.)
        __global__ void vertexShader(const Vertex3D* vertex3DPtr,
                                     unsigned int numVertices,
                                     PerspectiveProjMatrix ppm,
                                     Vertex2D* vertex2DPtr)
        {
            const unsigned int i = threadIdx.x;

            if (i >= numVertices) { return; }

            vertex2DPtr[i] = applyPerspective(vertex3DPtr[i], ppm);
        }

        __global__ void geometryShader(triangleIndices_t* indexPtr,
                                       unsigned int numTriangles,
                                       const Vertex2D* vertex2DPtr,
                                       const Vertex3D* vertex3DPtr,
                                       float* cross2DPtr,
                                       Vector3<float>* normalPtr,
                                       boundingBox_t* boundingBoxPtr)
        {
            const unsigned int i = threadIdx.x;

            if (i >= numTriangles) { return; }

            triangle2D_t triangle2D = getTriangle2D(indexPtr[i], vertex2DPtr);
            triangle3D_t triangle3D = getTriangle3D(indexPtr[i], vertex3DPtr);

            float cross = cross2D(triangle2D.p0, triangle2D.p2, triangle2D.p1);

            // In other functions the triangles' vertices are assumed to be in a clockwise order.
            if (cross < 0.f) {
                swap(triangle2D.p0, triangle2D.p1);
                swap(triangle3D.p0, triangle3D.p1);
                swap(indexPtr[i].p0, indexPtr[i].p1);
                cross *= -1;
            }

            cross2DPtr[i] = cross;
            normalPtr[i] = triangleNormal(triangle3D);
            boundingBoxPtr[i] = findBounds(triangle2D);
        }

        __global__ void fragmentShader(unsigned int rows,
                                       unsigned int cols,
                                       const triangleIndices_t* indexPtr,
                                       unsigned int numTriangles,
                                       const Vertex2D* vertex2DPtr,
                                       const Vertex3D* vertex3DPtr,
                                       const float* cross2DPtr,
                                       const Vector3<float>* normalPtr,
                                       const boundingBox_t* boundingBoxPtr,
                                       lightSource_t ls,
                                       char* characters)
        {
            const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
            const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= rows || x >= cols) { return; }

            fragment_t fragment;

            for (unsigned int k = 0; k < numTriangles; ++k) {
                triangle2D_t triangle2D = getTriangle2D(indexPtr[k], vertex2DPtr);
                triangle3D_t triangle3D = getTriangle3D(indexPtr[k], vertex3DPtr);
                float triangleArea2x = cross2DPtr[k];
                Vector3<float> normal = normalPtr[k];
                boundingBox_t boundingBox = boundingBoxPtr[k];
                barycentricCoords_t bc;

                if (triangleArea2x == 0.f) { continue; }

                if (inTriangle(triangle2D,
                               boundingBox,
                               Vertex2D({static_cast<float>(x), static_cast<float>(y)}),
                               triangleArea2x,
                               bc)) {
                    Vertex3D projectedPoint = getCartesianCoords(triangle3D, bc);

                    if (!fragment.occupied || fragment.coords.z() > projectedPoint.z()) {
                        fragment.occupied = true;
                        fragment.coords = projectedPoint;
                        fragment.normal = normal;
                    }
                }
            }

            if (fragment.occupied) {
                characters[y * cols + x] = brightnessToASCII(pointBrightness(fragment, ls));
            }
            else { characters[y * cols + x] = brightnessToASCII(0.f); }
        }

        /* Host auxiliary functions */
        [[nodiscard]] __host__ PerspectiveProjMatrix CCV2ScreenPPM(unsigned int screenRows, unsigned int screenCols)
        {
            return {PerspectiveMatrix(PM_NEAR_PLANE, PM_FAR_PLANE),
                    OrtProjMatrix(CCV_MIN_CORNER,
                                  CCV_MAX_CORNER,
                                  {0.f, 0.f, 0.f},
                                  {static_cast<float>(screenCols), static_cast<float>(screenRows), 0.f})};
        }

        __host__ void resizeSceneDependentPtrs(unsigned int numVertices, unsigned int numTriangles)
        {
            g_hostVertex3DDevPtr.resizeAndDiscardData(numVertices);
            g_hostVertex2DDevPtr.resizeAndDiscardData(numVertices);

            g_hostIndexDevPtr.resizeAndDiscardData(numTriangles);
            g_hostBoundingBoxDevPtr.resizeAndDiscardData(numTriangles);
            g_hostTriangleCross2DDevPtr.resizeAndDiscardData(numTriangles);
            g_hostTriangleNormalDevPtr.resizeAndDiscardData(numTriangles);
        }

        __host__ void resizeScreenDependentPtrs(unsigned int windowRows, unsigned int windowCols)
        {
            g_hostCharHostPtr.resizeAndDiscardData(windowRows * windowCols);
            g_hostCharDevPtr.resizeAndDiscardData(windowRows * windowCols);
        }

        __host__ void callVertexShader(const PerspectiveProjMatrix& PPM)
        {
            unsigned int numVertices = g_hostVertex3DDevPtr.size();

            unsigned int threadsPerBlock = std::min(numVertices, static_cast<unsigned int>(MAX_THREADS_PER_BLOCK));
            unsigned int numBlocks = (numVertices + threadsPerBlock - 1) / threadsPerBlock;

            vertexShader<<<numBlocks, threadsPerBlock>>>(g_hostVertex3DDevPtr.get(),
                                                         numVertices,
                                                         PPM,
                                                         g_hostVertex2DDevPtr.get());
            CUDA_CHECK(hipGetLastError());
        }

        __host__ void callGeometryShader()
        {
            unsigned int numTriangles = g_hostIndexDevPtr.size();

            unsigned int threadsPerBlock = std::min(numTriangles, static_cast<unsigned int>(MAX_THREADS_PER_BLOCK));
            unsigned int numBlocks = (numTriangles + threadsPerBlock - 1) / threadsPerBlock;

            geometryShader<<<numBlocks, threadsPerBlock>>>(g_hostIndexDevPtr.get(),
                                                           numTriangles,
                                                           g_hostVertex2DDevPtr.get(),
                                                           g_hostVertex3DDevPtr.get(),
                                                           g_hostTriangleCross2DDevPtr.get(),
                                                           g_hostTriangleNormalDevPtr.get(),
                                                           g_hostBoundingBoxDevPtr.get());
            CUDA_CHECK(hipGetLastError());
        }

        __host__ void callFragmentShader(const lightSource_t& ls,
                                         unsigned int windowRows,
                                         unsigned int windowCols)
        {
            unsigned int numTriangles = g_hostIndexDevPtr.size();

            dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
            dim3 numBlocks((windowCols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (windowRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

            fragmentShader<<<numBlocks, threadsPerBlock>>>(windowRows,
                                                           windowCols,
                                                           g_hostIndexDevPtr.get(),
                                                           numTriangles,
                                                           g_hostVertex2DDevPtr.get(),
                                                           g_hostVertex3DDevPtr.get(),
                                                           g_hostTriangleCross2DDevPtr.get(),
                                                           g_hostTriangleNormalDevPtr.get(),
                                                           g_hostBoundingBoxDevPtr.get(),
                                                           ls,
                                                           g_hostCharDevPtr.get());
            CUDA_CHECK(hipGetLastError());
        }

    } // anonymous

    __host__ void loadScene(const Scene& scene)
    {
        resizeSceneDependentPtrs(scene.verticesPtr().size(), scene.indicesPtr().size());

        scene.verticesPtr().loadToDev(g_hostVertex3DDevPtr.get());
        scene.indicesPtr().loadToDev(g_hostIndexDevPtr.get());
    }

    __host__ void draw(const lightSource_t& ls)
    {
        Vector2<unsigned int> windowDim = g_hostInactiveBuf.getWindowDimensions();

        windowDim.x() = std::min(windowDim.x(), windowDim.y());
        windowDim.y() = std::min(windowDim.x(), windowDim.y());

        PerspectiveProjMatrix PPM = CCV2ScreenPPM(windowDim.y(), windowDim.x());

        resizeScreenDependentPtrs(windowDim.y(), windowDim.x());

        callVertexShader(PPM);

        CUDA_CHECK(hipDeviceSynchronize());

        callGeometryShader();

        CUDA_CHECK(hipDeviceSynchronize());

        callFragmentShader(ls, windowDim.y(), windowDim.x());

        g_hostCharDevPtr.loadToHost(g_hostCharHostPtr.get());
        g_hostInactiveBuf.draw(g_hostCharHostPtr.get(), windowDim.y(), windowDim.x());
        g_hostInactiveBuf.activate();
        std::swap(g_hostActiveBuf, g_hostInactiveBuf);
    }

} // Custosh::Renderer